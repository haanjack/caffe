#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/upsample_layer.hpp"

namespace caffe {

template <typename Ftype, typename Btype>
__global__ void UpsampleForward(const int nthreads, int in_w, int in_h,
    int out_w, int out_h, const Ftype* bottom_data,
    const Ftype* bottom_mask, Ftype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int offset = index / (in_w * in_h) * out_w * out_h;
    int upsample_idx = static_cast<int>(bottom_mask[index]);
    top_data[offset + upsample_idx] = bottom_data[index];
  }
}

template <typename Ftype, typename Btype>
void UpsampleLayer<Ftype, Btype>::Forward_gpu(const vector<Blob*>& bottom,
      const vector<Blob*>& top) {
  const Ftype* bottom_data = bottom[0]->gpu_data();
  const Ftype* bottom_mask = bottom[1]->gpu_data();
  Ftype* top_data = top[0]->mutable_gpu_data();
  caffe_gpu_set(top[0]->count(), Ftype(0), top_data);
  int bottom_count = bottom[0]->count();
  UpsampleForward<Ftype><<<CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS>>>(
      bottom_count, bottom[0]->width(), bottom[0]->height(), 
      top[0]->width(), top[0]->height(), bottom_data, bottom_mask, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Ftype, typename Btype>
  __global__ void UpsampleBackward(const int nthreads, int in_w, int in_h,
      int out_w, int out_h, const Btype* top_diff,
      const Btype* bottom_mask, Btype* bottom_diff) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      int offset = index / (in_w * in_h) * out_w * out_h;
      int upsample_idx = static_cast<int>(bottom_mask[index]);
      bottom_diff[index] = top_diff[offset + upsample_idx];
    }
  }

template <typename Ftype, typename Btype>
void UpsampleLayer<Ftype, Btype>::Backward_gpu(const vector<Blob*>& top,
      const vector<bool>& propagate_down, const vector<Blob*>& bottom) {
  if (propagate_down[0]) {
    const Btype* top_diff = top[0]->gpu_diff();
    const Btype* bottom_mask = bottom[1]->gpu_data();
    Btype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int bottom_count = bottom[0]->count();
    caffe_gpu_set(bottom_count, Btype(0.), bottom_diff);
    UpsampleBackward<Btype><<<CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS>>>(
        bottom_count, bottom[0]->width(), bottom[0]->height(), 
        top[0]->width(), top[0]->height(), top_diff, bottom_mask, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS_FB(UpsampleLayer);


}  // namespace caffe
